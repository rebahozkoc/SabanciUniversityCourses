#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <string>

#include <fstream>
#include <sstream>
#include <vector>

#include <array>
#include <omp.h>
#include <stack>


using namespace std;


enum direction {
    d_down,
    d_right,
    none
};

#define COORD std::pair<int, int>

//#define DEBUG

int iter = 0;

/// Auxiliary functions

void display_arr(int* arr, int n) {
    cout << "arr: ";
    for (int i = 0; i < n; i++) {
        cout << arr[i] << " ";
    }
    cout << endl;
}

void print_coords(COORD start, COORD end) {

    cout << "Start:" << start.first << "," << start.second << endl;
    cout << "End:" << end.first << "," << end.second << endl;
}

int find_length(COORD start, COORD end, direction dir) {
    if (dir == d_down)
        return end.first - start.first;
    if (dir == d_right)
        return end.second - start.second;
    return -1;
}

void convert_sol(int** mat, int**& sol_mat, int m, int n) {

    sol_mat = new int* [m]; // Rows
    for (int i = 0; i < m; i++) {
        sol_mat[i] = new int[n]; // Cols
    }

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mat[i][j] == -2)
                sol_mat[i][j] = -2; // Empty value cell
            else
                sol_mat[i][j] = -1; // Hint or empty cell
        }
    }
}

void print_one_matrix(int** matrix, int m, int n) {
    std::cout << "Matrix: " << std::endl;
    for (int i = 0; i < m; i++) { // rows
        for (int j = 0; j < n; j++) { // cols
            std::cout << matrix[i][j] << "\t";
        }
        std::cout << "\n";
    }
}

void sol_to_file(int** mat, int** sol_mat, int m, int n, string fname) {
    ofstream to_write(fname);

    to_write << m << " " << n << "\n";

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            if (mat[i][j] != -2)
                to_write << mat[i][j] << " ";
            else
                to_write << sol_mat[i][j] << " ";
        }
        to_write << "\n";
    }

    to_write.close();
}

void read_matrix(int**& matrix, std::ifstream& afile, int m, int n) {

    matrix = new int* [m]; // rows

    for (int i = 0; i < m; i++) {
        matrix[i] = new int[n]; // cols
    }

    int val;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            afile >> val;
            matrix[i][j] = val;
        }
    }
}

/// Auxiliary functions

struct sum {
    COORD start;
    COORD end;

    int hint;
    int dir;
    int length;
    int posMin;
    int posMax;

    void print_sum() {
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << endl;
        cout << "############################" << endl;
    }

    sum(COORD _start, COORD _end, int _hint, direction _dir) : start(_start), end(_end), hint(_hint), dir(_dir) {
        length = find_length(_start, _end, _dir);
        // This is equal to hint - sum of numbers 9 + 8 + 7
        posMin = hint - 45 + ((8 - length) * (9 - length)) / 2;

        posMax = hint - (length * (length - 1)) / 2;

#ifdef DEBUG
        cout << "############################" << endl;
        cout << "Creating sum with: " << endl;
        print_coords(start, end);
        cout << "Hint: " << hint << endl;
        cout << "Direction: " << dir << endl;
        cout << "Length: " << length << endl;
        cout << "############################" << endl;
#endif
    }
};

COORD find_end(int** matrix, int m, int n, int i, int j, direction dir) { // 0 down 1 right

    if (dir == d_right) {
        for (int jj = j + 1; jj < n; jj++) {
            if (matrix[i][jj] != -2 || jj == n - 1) {
                if (matrix[i][jj] == -2 && jj == n - 1)
                    jj++;
                COORD END = COORD(i, jj);
                return END;
            }
        }
    }

    if (dir == d_down) {
        for (int ii = i + 1; ii < m; ii++) {
            if (matrix[ii][j] != -2 || ii == m - 1) {
                if (matrix[ii][j] == -2 && ii == m - 1)
                    ii++;
                COORD END = COORD(ii, j);
                return END;
            }
        }
    }

    cout << "ERROR: Find end is called with faulty parameters." << endl;
    return COORD(0, 0);
}

vector<sum> get_sums(int** matrix, int m, int n) {

    vector<sum> sums;

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int val = matrix[i][j];
            if (val != -1 && val != -2) {
                int hint = val;
                hint = hint / 10;
                // right sum
                if ((hint % 100) == 0) {
                    hint = (int)(hint / 100);
                    COORD START = COORD(i, j + 1);
                    COORD END = find_end(matrix, m, n, i, j, d_right);
                    sum _sum = sum(START, END, hint, d_right);
                    sums.push_back(_sum);
                }

                else {
                    int div = (int)(hint / 100);
                    int rem = (int)(hint % 100);
                    // down sum
                    if (div == 0 && rem != 0) {
                        COORD START = COORD(i + 1, j);
                        COORD END = find_end(matrix, m, n, i, j, d_down);
                        sum _sum = sum(START, END, rem, d_down);
                        sums.push_back(_sum);
                    }
                    // combined sum
                    if (div != 0 && rem != 0) {
                        COORD START1 = COORD(i + 1, j);
                        COORD START2 = COORD(i, j + 1);
                        COORD END1 = find_end(matrix, m, n, i, j, d_down);
                        COORD END2 = find_end(matrix, m, n, i, j, d_right);
                        sum _sum1 = sum(START1, END1, rem, d_down);
                        sum _sum2 = sum(START2, END2, div, d_right);
                        sums.push_back(_sum1);
                        sums.push_back(_sum2);
                    }
                }
            }
        }
    }
    return sums;
}



void print_flattened(int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int no_sums) {

    cout << "###h_sum_starts_x: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_starts_x[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_starts_y: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_starts_y[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_ends_x: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_ends_x[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_ends_y: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_ends_y[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_hints: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_hints[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_lengths: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_lengths[i] << " ";
    }
    cout << endl;

    cout << "###h_sum_dirs: " << endl;
    for (int i = 0; i < no_sums; i++) {
        cout << h_sum_dirs[i] << " ";
    }
    cout << endl;
}

void flatten_sol_mat(int** sol_mat, int* h_sol_mat, int m, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            h_sol_mat[i * n + j] = sol_mat[i][j];
        }
    }
}

void print_flattened_matrix(int* h_sol_mat, int m, int n) {

    cout << "###Flattened matrix: " << endl;
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            cout << h_sol_mat[i * n + j] << " ";
        }
        cout << endl;
    }
    cout << endl;
}


__global__
void print_flattened_matrix_device(int* d_sol_mat, int m, int n) {

    printf("###Flattened matrix: %d, %d\n", m, n);
    
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", d_sol_mat[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

__device__
bool checkSumPartial(int* mat, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    int sum = 0;
    if (sum_dir == 1) { // right
        for (int j = sum_start_y; j < sum_end_y; j++) {
            if (mat[sum_start_x * n + j] != -2) {
                sum += mat[sum_start_x *n + j];
            }

        }
    }
    else { // down
        for (int i = sum_start_x; i < sum_end_x; i++) {
            if (mat[i*n + sum_start_y] != -2) {
                sum += mat[i * n + sum_start_y];
            }
        }
    }

    if (sum > sum_hint) {
        return false;
    }
    return true;
}
__device__
bool checkSumComplete(int* mat, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    int sum = 0;
    if (sum_dir == 1) { // right
        for (int j = sum_start_y; j < sum_end_y; j++) {
            if (mat[sum_start_x * n + j] != -2) {
                sum += mat[sum_start_x * n + j];
            }

        }
    }
    else { // down
        for (int i = sum_start_x; i < sum_end_x; i++) {
            if (mat[i* n + sum_start_y] != -2) {
                sum += mat[i * n + sum_start_y];
            }
        }
    }
    if (sum != sum_hint) {
        return false;
    }
    return true;
}

__device__
bool areElementsUniqueExceptEmpties(int* mat, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    if (sum_dir == 1) { // right
        for (int j = sum_start_y; j < sum_end_y; j++) {
            if (mat[sum_start_x * n + j] != -2) {
                for (int k = j + 1; k < sum_end_y; k++) {
                    if (mat[sum_start_x * n +j] == mat[sum_start_x * n + k] && mat[sum_start_x * n + k] != -2) {
                        // If the same non-empty element is found, it means it's a duplicate, and we can return false
                        return false;
                    }
                }
            }
        }
    }
    else { // down
        for (int i = sum_start_x; i < sum_end_x; i++) {
            if (mat[i* n + sum_start_y] != -2) {
                for (int k = i + 1; k < sum_end_x; k++) {
                    if (mat[i* n + sum_start_y] == mat[k* n + sum_start_y] && mat[k * n + sum_start_y] != -2) {
                        return false;
                    }
                }
            }
        }
    }
    return true;
}

__device__
bool areElementsUnique(int* mat, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    if (sum_dir == 1) { // right
        for (int j = sum_start_y; j < sum_end_y; j++) {
            for (int k = j + 1; k < sum_end_y; k++) {
                if (mat[sum_start_x * n + j] == mat[sum_start_x * n + k]) {
                    // If the same element is found, it means it's a duplicate, and we can return false
                    return false;
                }
            }
        }
    }
    else { // down
        for (int i = sum_start_x; i < sum_end_x; i++) {
            for (int k = i + 1; k < sum_end_x; k++) {
                if (mat[i* n + sum_start_y] == mat[k* n + sum_start_y]) {
                    return false;
                }
            }
        }
    }
    return true;
}
__device__
bool isArrFull(int* mat, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    if (sum_dir == 1) { // right
        for (int j = sum_start_y; j < sum_end_y; j++) {
            if (mat[sum_start_x * n + j] == -2)
                return false;
        }
    }
    else { // down
        for (int i = sum_start_x; i < sum_end_x; i++) {
            if (mat[i* n + sum_start_y] == -2)
                return false;
        }
    }
    return true;
}

__device__
bool isACandidate(int val, int posMin, int posMax){
    if (val < posMin) {
        return false;
    }
    if (val > posMax) {
        return false;
    }
    return true;
}


__device__
bool isLastCell(int curr_i, int curr_j, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir) {
    if (sum_dir == 1) { // to right
        if (sum_start_x == curr_i && sum_end_y - 1 == curr_j) {
            return true;
        }
    }
    else {
        if (sum_start_y == curr_j && sum_end_x - 1 == curr_i) {
            {
                return true;
            }
        }
    }
    return false;
}

__device__
bool fullCheck(int* mat, int curr_i, int curr_j, int val, int m, int n, int sum_start_x, int sum_start_y, int sum_end_x, int sum_end_y, int sum_hint, int sum_length, int sum_dir, int sum_min, int sum_max) {
    int i = curr_i;
    int j = curr_j;
    
    if (sum_dir == 1) { // right
        // current coordinate is not in the sum
        if (!((i == sum_start_x) && (j >= sum_start_x) && (j < sum_end_y))) {
            return true;
        }
    }
    else { // down
        if (!(j == sum_start_y && i >= sum_start_x && i < sum_end_x)) {
            return true;
        }
    }
    if (!isACandidate(val, sum_min, sum_max)) {
        return false;
    }
    if (!areElementsUniqueExceptEmpties(mat, m, n, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir)) {
        return false;
    }
    if (isArrFull(mat, m, n, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir) || isLastCell(curr_i, curr_j, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir)) {
        return checkSumComplete(mat, m, n, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir) && areElementsUnique(mat, m, n, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir);
    }
    else {
        return checkSumPartial(mat, m, n, sum_start_x, sum_start_y, sum_end_x, sum_end_y, sum_hint, sum_length, sum_dir);
    }
}


int mat_iter_get_next_i(int* mat, int m, int n, int curr_i, int curr_j) {
    do {
        if (curr_j + 1 < n) {
            curr_j += 1;
        }
        else {
            curr_i += 1;
            curr_j = 0;
        }
    } while (curr_i != m && (mat[curr_i * n + curr_j] != -2));
    if (curr_i == m) {
        curr_i = -999; // end of the iteration
        curr_j = -999;

    }
    return curr_i;
}


int mat_iter_get_next_j(int* mat, int m, int n, int curr_i, int curr_j) {
    do {
        if (curr_j + 1 < n) {
            curr_j += 1;
        }
        else {
            curr_i += 1;
            curr_j = 0;

        }
    } while (curr_i != m && (mat[curr_i * n + curr_j] != -2));
    if (curr_i == m) {
        curr_i = -999; // end of the iteration
        curr_j = -999;

    }
    return curr_j;
}

int mat_iter_init_i(int* mat, int m, int n) {
    int curr_i = 0;
    int curr_j = 0;
    if (mat[curr_i*n + curr_j] != -2) {
        curr_i = mat_iter_get_next_i(mat, m, n, curr_i, curr_j);
        curr_j = mat_iter_get_next_j(mat, m, n, curr_i, curr_j);
    }
    return curr_i;
}

int mat_iter_init_j(int* mat, int m, int n) {
    int curr_i = 0;
    int curr_j = 0;
    if (mat[curr_i * n + curr_j] != -2) {
        curr_i = mat_iter_get_next_i(mat, m, n, curr_i, curr_j);
        curr_j = mat_iter_get_next_j(mat, m, n, curr_i, curr_j);
    }
    return curr_j;
}

void flatten_sums(vector<sum> sums, int* h_sum_starts_x, int* h_sum_starts_y, int* h_sum_ends_x, int* h_sum_ends_y, int* h_sum_hints, int* h_sum_lengths, int* h_sum_dirs, int* h_sum_pos_mins, int* h_sum_pos_maxs,  int no_sums) {

    for (int i = 0; i < no_sums; i++) {

        h_sum_starts_x[i] = sums[i].start.first;
        h_sum_starts_y[i] = sums[i].start.second;

        h_sum_ends_x[i] = sums[i].end.first;
        h_sum_ends_y[i] = sums[i].end.second;

        h_sum_hints[i] = sums[i].hint;
        h_sum_lengths[i] = sums[i].length;

        h_sum_dirs[i] = sums[i].dir;
        
        h_sum_pos_mins[i] = sums[i].posMin;
        h_sum_pos_maxs[i] = sums[i].posMax;
    }
}

__global__
void full_check_kernel(int* mat, int curr_i, int curr_j, int val, int m, int n, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints,
    int* d_sum_lengths, int* d_sum_dirs, int * d_sum_mins, int * d_sum_maxs, int no_sums, volatile bool* partial_correct) {
    volatile __shared__ bool someoneFoundIt;

    int i = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (threadIdx.x == 0) someoneFoundIt = *partial_correct;
    __syncthreads();
    if (someoneFoundIt && i < no_sums) {
        bool iFoundItFalse = !fullCheck(mat, curr_i, curr_j, val, m, n,
            d_sum_starts_x[i],
            d_sum_starts_y[i],
            d_sum_ends_x[i],
            d_sum_ends_y[i],
            d_sum_hints[i],
            d_sum_lengths[i],
            d_sum_dirs[i],
            d_sum_mins[i],
            d_sum_maxs[i]);
        if (iFoundItFalse) { someoneFoundIt = false; *partial_correct = false; }
        if (threadIdx.x == 0 && !(*partial_correct)) someoneFoundIt = false;
    }
}
///////////////////
// CUDA FUNCTIONS //
///////////////////


bool solution(int* h_sol_mat, int* d_sol_mat, int m, int n, stack<int> iter_i_stack, stack<int> iter_j_stack, stack<int> val_stack, int* d_sum_starts_x, int* d_sum_starts_y, int* d_sum_ends_x, int* d_sum_ends_y, int* d_sum_hints,
    int* d_sum_lengths, int* d_sum_dirs, int* d_sum_mins, int* d_sum_maxs, int no_sums) {

    const int GRIDSIZE = (no_sums + 1023) / 1024;
    const int THREADSIZE = min(1024, no_sums);
    bool* partial_correctness = (bool*)malloc(sizeof(bool));
    
    bool* d_partial_correctness;
    hipMalloc(&d_partial_correctness, sizeof(bool));
    while (!iter_i_stack.empty()) {
        int iter_i = iter_i_stack.top();
        int iter_j = iter_j_stack.top();
        int curr_val = val_stack.top();

        iter_i_stack.pop();
        iter_j_stack.pop();
        val_stack.pop();

        if (curr_val < 10) {
            h_sol_mat[iter_i *n + iter_j] = curr_val;

            * partial_correctness = true;
            hipMemcpy(d_partial_correctness, partial_correctness, sizeof(bool), hipMemcpyHostToDevice);
            
            hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            
            full_check_kernel << < GRIDSIZE, THREADSIZE >> > (d_sol_mat, iter_i, iter_j, curr_val, m, n, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
                d_sum_lengths, d_sum_dirs, d_sum_mins, d_sum_maxs, no_sums, d_partial_correctness);
            hipDeviceSynchronize();
            hipMemcpy(partial_correctness, d_partial_correctness, sizeof(bool), hipMemcpyDeviceToHost);
            if (*partial_correctness) {
                iter_i_stack.push(iter_i);
                iter_j_stack.push(iter_j);
                val_stack.push(curr_val);

                int iter_i_next = mat_iter_get_next_i(h_sol_mat, m, n, iter_i_stack.top(), iter_j_stack.top());
                int iter_j_next = mat_iter_get_next_j(h_sol_mat, m, n, iter_i_stack.top(), iter_j_stack.top());

                if (iter_i_next == -999 || iter_j_next == -999) {
                    cout << "END INSIDE:" << endl;
                    hipMemcpy(h_sol_mat, d_sol_mat, (m * n) * sizeof(int), hipMemcpyDeviceToHost);
                    print_flattened_matrix(h_sol_mat, m, n);
                    return true;
                }
                //state_stack.push(state(iter_i_next, iter_j_next, 1));
                iter_i_stack.push(iter_i_next);
                iter_j_stack.push(iter_j_next);
                val_stack.push(1);
            }
            else {
                curr_val += 1;
                h_sol_mat[iter_i * n + iter_j] = -2;
                hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);
      
                iter_i_stack.push(iter_i);
                iter_j_stack.push(iter_j);
                val_stack.push(curr_val);
            }

        }
        else {
            while (!iter_i_stack.empty()) {
                int iter_i = iter_i_stack.top();
                int iter_j = iter_j_stack.top();
                int curr_val = val_stack.top();

                iter_i_stack.pop();
                iter_j_stack.pop();
                val_stack.pop();
                h_sol_mat[iter_i * n+ iter_j] = -2;
                hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);
                
                curr_val += 1;
                if (curr_val < 10) {
                    iter_i_stack.push(iter_i);
                    iter_j_stack.push(iter_j);
                    val_stack.push(curr_val);
                    break;
                }
            }
        }
    }
    return false;
}


int main(int argc, char** argv) {

    std::string filename(argv[1]);
    std::ifstream file;
    file.open(filename.c_str());

    int m, n;

    file >> m;
    file >> n;

    int** mat;
    read_matrix(mat, file, m, n);
    print_one_matrix(mat, m, n);

    int** sol_mat;
    convert_sol(mat, sol_mat, m, n);
    print_one_matrix(sol_mat, m, n);

    double start;
    double end;

    // CUDA
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("==prop== Running on device: %d -- %s \n", 0, prop.name);
    printf("==prop== #of SM -- %d \n", prop.multiProcessorCount);
    printf("==prop== Max Threads Per Block: -- %d \n", prop.maxThreadsPerBlock);

    vector<sum> sums = get_sums(mat, m, n);

    int no_sums = sums.size();

    // Flattening sums and matrix
    int* h_sum_starts_x = new int[no_sums];
    int* h_sum_starts_y = new int[no_sums];
    int* h_sum_ends_x = new int[no_sums];
    int* h_sum_ends_y = new int[no_sums];
    int* h_sum_hints = new int[no_sums];
    int* h_sum_lengths = new int[no_sums];
    int* h_sum_dirs = new int[no_sums];
    int* h_sum_mins = new int[no_sums];
    int* h_sum_maxs = new int[no_sums];

    flatten_sums(sums, h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, h_sum_mins, h_sum_maxs, no_sums);

    print_flattened(h_sum_starts_x, h_sum_starts_y, h_sum_ends_x, h_sum_ends_y, h_sum_hints, h_sum_lengths, h_sum_dirs, no_sums);

    int* h_sol_mat;
    h_sol_mat = new int[m * n];
    flatten_sol_mat(sol_mat, h_sol_mat, m, n);

    print_flattened_matrix(h_sol_mat, m, n);

    // Declare device pointers and copy data into device
    int* d_sum_starts_x, * d_sum_starts_y, * d_sum_ends_x, * d_sum_ends_y, * d_sum_hints, * d_sum_lengths, * d_sum_dirs, * d_sum_mins, * d_sum_maxs, * d_sol_mat;

    hipMalloc(&d_sum_starts_x, no_sums * sizeof(int));
    hipMalloc(&d_sum_starts_y, no_sums * sizeof(int));
    hipMalloc(&d_sum_ends_x, no_sums * sizeof(int));
    hipMalloc(&d_sum_ends_y, no_sums * sizeof(int));
    hipMalloc(&d_sum_hints, no_sums * sizeof(int));
    hipMalloc(&d_sum_lengths, no_sums * sizeof(int));
    hipMalloc(&d_sum_dirs, no_sums * sizeof(int));
    hipMalloc(&d_sum_mins, no_sums * sizeof(int));
    hipMalloc(&d_sum_maxs, no_sums * sizeof(int));

    hipMalloc(&d_sol_mat, (m * n) * sizeof(int));
    
    hipMemcpy(d_sum_starts_x, h_sum_starts_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_starts_y, h_sum_starts_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_ends_x, h_sum_ends_x, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_ends_y, h_sum_ends_y, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_hints, h_sum_hints, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_lengths, h_sum_lengths, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_dirs, h_sum_dirs, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_mins, h_sum_mins, no_sums * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sum_maxs, h_sum_maxs, no_sums * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_sol_mat, h_sol_mat, (m * n) * sizeof(int), hipMemcpyHostToDevice);

    int iter_i = mat_iter_init_i(h_sol_mat, m, n);
    int iter_j = mat_iter_init_j(h_sol_mat, m, n);
    
    stack<int> iter_i_stack, iter_j_stack, val_stack;
    iter_i_stack.push(iter_i);
    iter_j_stack.push(iter_j);
    val_stack.push(1);

    start = omp_get_wtime();
    bool result = solution(h_sol_mat, d_sol_mat, m, n, iter_i_stack, iter_j_stack, val_stack, d_sum_starts_x, d_sum_starts_y, d_sum_ends_x, d_sum_ends_y, d_sum_hints,
        d_sum_lengths, d_sum_dirs, d_sum_mins, d_sum_maxs, no_sums);
    end = omp_get_wtime();
    printf("Work took %f seconds\n", end - start);

    //sol_to_file(mat, sol_mat, m, n, "solution.kakuro");
    if (result) {
        cout << "SUCCESS" << endl;
    }
    else {
        cout << "COULD NOT SOLVE" << endl;
    }
    hipDeviceSynchronize();
    // DELETE PART 
    for (int i = 0; i < m; i++) {
        delete mat[i];
        delete sol_mat[i];
    }

    delete mat;
    delete sol_mat;

    delete h_sum_starts_x;
    delete h_sum_starts_y;
    delete h_sum_ends_x;
    delete h_sum_ends_y;
    delete h_sum_hints;
    delete h_sum_lengths;
    delete h_sum_dirs;
    delete h_sol_mat;

    hipFree(d_sum_starts_x);
    hipFree(d_sum_starts_y);
    hipFree(d_sum_ends_x);
    hipFree(d_sum_ends_y);
    hipFree(d_sum_hints);
    hipFree(d_sum_lengths);
    hipFree(d_sum_dirs);
    hipFree(d_sol_mat);
    hipFree(d_sum_mins);
    hipFree(d_sum_maxs);

    return 0;
}
